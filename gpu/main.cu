
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <cmath>
#include <limits>
#include <float.h>
#include <string>

const int map_width = 6000;
const int range = 15; // the width of the box shaped range one can see from the origin
const int range_radius = (range - 1) / 2;
const char* dem_location = "../data/srtm_14_04_6000x6000_short16.raw";
const char* output_location = "./viewshed-gpu.raw";

__host__ int openDem(const char* location, std::vector<short> &dem);
__host__ int writeViewshed(const char* location, std::vector<short> &vshed);
__host__ void printRows(std::vector<short> &rows);
__device__ void bLineDown(int x0, int y0, int x1, int y1, short* dem_d, float& max_slope, short& origin_height, int& map_width_d);
__device__ void bLineUp(int x0, int y0, int x1, int y1, short* dem_d, float& max_slope, short& origin_height, int& map_width_d);
__device__ void bLine(int x0, int y0, int x1, int y1, short* dem_d, float& max_slope, short& origin_height, int& map_width_d);
__global__ void singleViewshedCount(short* dem_d, short* vshed_d, int map_width_d, int range_radius_d);
__device__ void toGridCoords(int& index, int& x, int& y, int& map_width_d);
__device__ void toFlatCoords(int& x, int& y, int& index, int& map_width_d);
__host__ void compareOutput(std::string filename1, std::string filename2);
__host__ void printRowsPrim(short* rows);
__host__ int writeViewshedPrim(const char* location, short* &vshed);

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

int main()
{
    std::vector<short> dem;
    if (openDem(dem_location, dem))
    {
        printf("Failed to open input file: %s\n", dem_location);
        return 1; // File failed to open, exit program
    }
    printf("Opened input file: %s\nVector size: %d\n", dem_location, dem.size());

    // small test dem
    // dem = {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1,
    //        1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1};

    short* vshed = new short[dem.size()];

    // Allocate memory on device
    short *dem_d, *vshed_d;
    hipMalloc(&dem_d, dem.size() * sizeof(short));
    hipMalloc(&vshed_d, dem.size() * sizeof(short));
    hipError_t error = hipMemcpy(dem_d, &dem[0], dem.size() * sizeof(short), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("Host to device failed.\n");
        return 1;
    }

    // Execute kernel
    int tpb = 600;
    int bpg = ((map_width * map_width) / tpb) + 1;
    dim3 dimGrid(bpg, 1, 1);
    dim3 dimBlock(tpb, 1, 1);

    printf("tpb: %d\nbpg: %d\n", tpb, bpg);
    printf("Launching the kernel...");
    singleViewshedCount<<<dimGrid, dimBlock>>>(dem_d, vshed_d, map_width, range_radius);
    // singleViewshedCount<<<12, 12>>>(dem_d, vshed_d, map_width, range_radius);

    hipDeviceSynchronize();
    printf("Kernel done!\n");

    // Copy array from device to host
    error = hipMemcpy(vshed, vshed_d, dem.size() * sizeof(short), hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        printf("Device to host failed.\n");
        return 1;
    }

    // Output
    // writeViewshed(output_location, vshed);
    
    // printRows(vshed);
    writeViewshedPrim(output_location, vshed);
    // printRowsPrim(vshed);

    // compareOutput("./viewshed-gpu.raw", "../distributed/viewshed-64.raw");

    // Free device memory
    hipFree(dem_d);
    hipFree(vshed_d);
}

/*
    Compare two RAW files and count differences
*/
__host__ void compareOutput(std::string filename1, std::string filename2)
{
    std::ifstream file1(filename1, std::ios::binary);
    std::ifstream file2(filename2, std::ios::binary);

    if (!file1.is_open() || !file2.is_open())
    {
        printf("Failed to open one of two files\n");
        return;
    }

    short buf1, buf2;
    file1.seekg(0, std::ios::beg);
    file2.seekg(0, std::ios::beg);
    int count = 0;
    while (file1.read((char*)&buf1, sizeof(short)) && file2.read((char*)&buf2, sizeof(short)))
    {
        if (buf1 != buf2)
        {
            ++count;
        }
    }

    printf("Number of errors: %d\n", count);
}

/*
    Print contents of vector by row
*/
__host__ void printRows(std::vector<short> &rows)
{
    for (std::size_t row = 0; row < rows.size() / map_width; ++row)
    {
        for (int cell = 0; cell < map_width; ++cell)
        {
            printf("%4d", rows[row * map_width + cell]);
        }
        printf("\n");
    }
}

/*
    Print contents of array by row
*/
__host__ void printRowsPrim(short* rows)
{
    for (int row = 0; row < map_width * map_width / map_width; ++row)
    {
        for (int cell = 0; cell < map_width; ++cell)
        {
            printf("%4d", rows[row * map_width + cell]);
        }
        printf("\n");
    }
}

/*
    Open DEM from RAW file and store in 1-dimensional short array
*/
__host__ int openDem(const char* location, std::vector<short> &dem)
{
    std::ifstream input_raw(location, std::ios::binary);
    if (!input_raw.is_open())
    {
        return 1;
    }

    int i = 0;
    short buf;
    input_raw.seekg(0, std::ios::beg);
    while (input_raw.read((char *)&buf, sizeof(short)))
    {
        dem.push_back(buf);
        ++i;
    }

    input_raw.close();
    return 0;
}

/*
    Write computed viewshed counts to raw file
*/
__host__ int writeViewshed(const char* location, std::vector<short> &vshed)
{
    std::ofstream output_raw(location, std::ios::binary | std::ios::trunc);
    if (!output_raw.is_open())
    {
        return 1;
    }

    printf("Writing output to file %s...  ", location);
    output_raw.write((char const*)&vshed[0], map_width * map_width * sizeof(short));
    printf("Done!\n");

    output_raw.close();
    return 0;
}

/*
    Write computed viewshed counts to raw file
*/
__host__ int writeViewshedPrim(const char* location, short* &vshed)
{
    std::ofstream output_raw(location, std::ios::binary | std::ios::trunc);
    if (!output_raw.is_open())
    {
        return 1;
    }

    printf("Writing output to file %s...  ", location);
    output_raw.write((char*)vshed, map_width * map_width * sizeof(short));
    printf("Done!\n");

    output_raw.close();
    return 0;
}

/*
    Calculate the viewshed for a given origin
*/
__global__ void singleViewshedCount(short* dem_d, short* vshed_d, int map_width_d, int range_radius_d)
{
    int origin = blockIdx.x * blockDim.x + threadIdx.x;
    int map_size = map_width_d * map_width_d;
    if (origin >= map_size) return; // check threads out of bounds
    short count = 0;
    short origin_height = dem_d[origin];
    short p_height;
    float dx, dy, d, slope, ox, oy, px, py;
    ox = static_cast<float>(origin % map_width_d);
    oy = static_cast<float>(origin / map_width_d);

    int p;
    for (int i = -range_radius_d; i <= range_radius_d; ++i)
    {
        for (int j = -range_radius_d; j <= range_radius_d; ++j)
        {
            p = origin + (i * map_width_d + j);
            if (p < 0 || p > map_size || p == origin) continue; // top/bottom bounds and self
            p_height = dem_d[p];
            px = static_cast<float>(p % map_width_d);
            py = static_cast<float>(p / map_width_d);
            dx = ox - px;
            dy = oy - py;
            if ((j < 0 && dx < -j) || (j > 0 && dx > j)) continue; // left/right bounds
            d = hypot(dx, dy);
            slope = (p_height - origin_height) / d;
            float max_slope = -FLT_MAX;
            bLine(ox, oy, px, py, dem_d, max_slope, origin_height, map_width_d);
            if (slope >= max_slope)
            {
                // printf("I see %d(%d) from %d(%d). Slope is %f\n", p, p_height, origin, origin_height, slope);
                ++count;
            }
        }
    }

    vshed_d[origin] = count;
}

/*
    Convert 1-dimensional index to 2-dimensional indices
*/
__device__ void toGridCoords(int& index, int& x, int& y, int& map_width_d)
{
    x = index % map_width_d;
    y = index / map_width_d;
}

/*
    Convert 2-dimensional indices to 1-dimensional index
*/
__device__ void toFlatCoords(int& x, int& y, int& index, int& map_width_d)
{
    index = (y * map_width_d) + x;
}

/*
    Calculate line when the slope is negative (or 0!!)
    Source: https://en.wikipedia.org/wiki/Bresenham%27s_line_algorithm
*/
__device__ void bLineDown(int x0, int y0, int x1, int y1, short* dem_d, float& max_slope, short& origin_height, int& map_width_d)
{
    float diffx, diffy, diff;
    float slope = -FLT_MAX;
    int m;
    int dx = x1 - x0;
    int dy = y1 - y0;
    int yi = 1;
    if (dy < 0)
    {
        yi = -1;
        dy = -dy;
    }
    int D = (2 * dy) - dx;
    int y = y0;

    for (int x = x0 + 1; x < x1; ++x)
    {
        // flatten x,y coordinates to 1D -> m
        toFlatCoords(x, y, m, map_width_d);
        diffx = x - x0;
        diffy = y - y0;
        diff = hypot(diffx, diffy);
        slope = (dem_d[m] - origin_height) / diff;
        if (slope > max_slope) max_slope = slope;
        if (D > 0)
        {
            y += yi;
            D += (2 * (dy - dx));
        }
        else
        {
            D += 2 * dy;
        }
    }
}

/*
    Calculate line when the slope is positive
    Source: https://en.wikipedia.org/wiki/Bresenham%27s_line_algorithm
*/
__device__ void bLineUp(int x0, int y0, int x1, int y1, short* dem_d, float& max_slope, short& origin_height, int& map_width_d)
{
    float diffx, diffy, diff, slope;
    int m;
    int dx = x1 - x0;
    int dy = y1 - y0;
    int xi = 1;
    if (dx < 0)
    {
        xi = -1;
        dx = -dx;
    }
    int D = (2 * dx) - dy;
    int x = x0;

    for (int y = y0 + 1; y < y1; ++y)
    {
        toFlatCoords(x, y, m, map_width_d);
        diffx = x - x0;
        diffy = y - y0;
        diff = hypot(diffx, diffy);
        slope = (dem_d[m] - origin_height) / diff;
        if (slope > max_slope) max_slope = slope;
        if (D > 0)
        {
            x += xi;
            D += (2 * (dx - dy));
        }
        else
        {
            D += 2 * dx;
        }
    }
}

/*
    Calculate what points are intersected by a line between (x0,y0) and (x1,y1)
    Source: https://en.wikipedia.org/wiki/Bresenham%27s_line_algorithm
*/
__device__ void bLine(int x0, int y0, int x1, int y1, short* dem_d, float& max_slope, short& origin_height, int& map_width_d)
{
    if (abs(y1 - y0) < abs(x1 - x0))
    {
        (x0 > x1) ? bLineDown(x1, y1, x0, y0, dem_d, max_slope, origin_height, map_width_d) : bLineDown(x0, y0, x1, y1, dem_d, max_slope, origin_height, map_width_d);
    }
    else
    {
        (y0 > y1) ? bLineUp(x1, y1, x0, y0, dem_d, max_slope, origin_height, map_width_d) : bLineUp(x0, y0, x1, y1, dem_d, max_slope, origin_height, map_width_d);
    }
}